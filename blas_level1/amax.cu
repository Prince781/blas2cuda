#include "level1.h"

template <typename T, typename R>
static void _cblas_i_amax(const int n,
        const T *x,
        const int incx,
        R *result,
        hipblasStatus_t amax_func(hipblasHandle_t, int, 
            const T *, int,
            R *))
{
    const T *gpu_x;
    const int size_x = size(n, incx, sizeof(*x));
    const struct objinfo *x_info;

    gpu_x = (const T *) b2c_place_on_gpu((void *) x, size_x, &x_info, NULL);

    amax_func(b2c_handle, n, gpu_x, incx, result);

    if (hipPeekAtLastError() != hipSuccess)
        b2c_fatal_error(hipGetLastError(), __func__);

    b2c_cleanup_gpu_ptr((void *) gpu_x, x_info);
}

DECLARE_CBLAS_I_AMAX(s, float) {
    CBLAS_INDEX idx;
    _cblas_i_amax(n, x, incx, &idx, &hipblasIsamax);
    return idx;
}

DECLARE_CBLAS_I_AMAX(d, double) {
    CBLAS_INDEX idx;
    _cblas_i_amax(n, x, incx, &idx, &hipblasIdamax);
    return idx;

}

DECLARE_CBLAS_I_AMAX(c, float _Complex) {
    CBLAS_INDEX idx;
    _cblas_i_amax(n, (hipComplex *) x, incx, &idx, &hipblasIcamax);
    return idx;
}

DECLARE_CBLAS_I_AMAX(z, double _Complex) {
    CBLAS_INDEX idx;
    _cblas_i_amax(n, (hipDoubleComplex *) x, incx, &idx, &hipblasIzamax);
    return idx;
}

