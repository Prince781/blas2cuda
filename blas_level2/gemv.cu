#include "level2.h"

template <typename T>
void _cblas_gemv (const CBLAS_LAYOUT Layout,
        CBLAS_TRANSPOSE trans,
        const int m, const int n,
        const T alpha,
        const T *A, const int lda,
        const T *x, const int incx,
        const T beta,
        T *y, const int incy,
        hipblasStatus_t gemv_func(hipblasHandle_t,
            hipblasOperation_t,
            int, int, 
            const T *,
            const T *, int,
            const T *, int,
            const T *,
            T *, int),
        geam_t<T> geam_func)
{
    const T *gpu_A, *gpu_x;
    T *gpu_y;
    const int size_A = size(n, lda, sizeof(*A));
    const int size_x = size(n, incx, sizeof(*x));
    const int size_y = size(n, incy, sizeof(*y));
    const struct objinfo *A_info, *x_info, *y_info;
    int rows_A, cols_A;
    hipblasOperation_t op = (hipblasOperation_t) (trans - CblasNoTrans);

    if (Layout == CblasRowMajor && trans == CblasConjTrans) {
        /* create a new buffer that is the transpose matrix of A*/
        T *gpu_A_conj;

        A_info = NULL;
        rows_A = n;
        cols_A = m;
        gpu_A_conj = (T *) b2c_copy_to_gpu((void *) A, size_A);

        /* transpose A */
        geam_func(b2c_handle, HIPBLAS_OP_T, HIPBLAS_OP_N,
                rows_A, cols_A,
                &alpha,
                gpu_A_conj, lda,
                0,
                0, 0,
                gpu_A_conj, lda);

        if (hipPeekAtLastError() != hipSuccess)
            b2c_fatal_error(hipGetLastError(), __func__);

        gpu_A = gpu_A_conj;
    } else {
        gpu_A = (const T *) b2c_place_on_gpu((void *) A, size_A, &A_info, NULL);
        if (Layout == CblasRowMajor) {
            if (trans == CblasNoTrans)
                op = HIPBLAS_OP_T;
            else if (trans == CblasTrans)
                op = HIPBLAS_OP_N;
            rows_A = n;
            cols_A = m;
        } else {
            rows_A = m;
            cols_A = n;
        }
    }

    gpu_x = (const T *) b2c_place_on_gpu((void *) x, size_x, &x_info,
            (void *) gpu_A, A_info,
            NULL);
    gpu_y = (T *) b2c_place_on_gpu(NULL, size_y, &y_info,
            (void *) gpu_A, A_info,
            (void *) gpu_x, x_info,
            NULL);

    gemv_func(b2c_handle, op,
            rows_A, cols_A,
            &alpha, 
            gpu_A, lda,
            gpu_x, incx,
            &beta,
            gpu_y, incy);

    if (hipPeekAtLastError() != hipSuccess)
        b2c_fatal_error(hipGetLastError(), __func__);

    if (!y_info)
        b2c_copy_from_gpu(y, gpu_y, size_y);

    b2c_cleanup_gpu_ptr((void *) gpu_A, A_info);
    b2c_cleanup_gpu_ptr((void *) gpu_x, x_info);
    b2c_cleanup_gpu_ptr((void *) gpu_y, y_info);

}

DECLARE_CBLAS__GEMV(s, float) {
    _cblas_gemv(Layout, trans, 
            m, n, 
            alpha, 
            A, lda,
            x, incx,
            beta,
            y, incy, &hipblasSgemv, &hipblasSgeam);
}

DECLARE_CBLAS__GEMV(d, double) {
    _cblas_gemv(Layout, trans, 
            m, n, 
            alpha, 
            A, lda,
            x, incx,
            beta,
            y, incy, &hipblasDgemv, &hipblasDgeam);
}

DECLARE_CBLAS__GEMV(c, float _Complex) {
    _cblas_gemv(Layout, trans, 
            m, n, 
            *(hipComplex *) &alpha, 
            (hipComplex *) A, lda,
            (hipComplex *) x, incx,
            *(hipComplex *) &beta,
            (hipComplex *) y, incy, &hipblasCgemv, &hipblasCgeam);
}

DECLARE_CBLAS__GEMV(z, double _Complex) {
    _cblas_gemv(Layout, trans, 
            m, n, 
            *(hipDoubleComplex *) &alpha, 
            (hipDoubleComplex *) A, lda,
            (hipDoubleComplex *) x, incx,
            *(hipDoubleComplex *) &beta,
            (hipDoubleComplex *) y, incy, &hipblasZgemv, &hipblasZgeam);
}
