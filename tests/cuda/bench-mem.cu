// see https://devblogs.nvidia.com/parallelforall/how-optimize-data-transfers-cuda-cc/

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <assert.h>

__global__ void do_stuff(float *dev_A, size_t size) {
    /* blockDim.{x,y} is a constant = the dimension of the grid */
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < size) {
        dev_A[i] = 13;
    }
}

// Convenience function for checking CUDA runtime API results
// can be wrapped around any runtime API call. No-op in release builds.
inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != cudaSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", 
            cudaGetErrorString(result));
    if (result != cudaSuccess)
	abort();
  }
#endif
  return result;
}

void profileCopies(float        *h_a, 
                   float        *h_b, 
                   float        *d, 
                   unsigned int  n,
                   const char   *desc)
{
  printf("\n%s transfers\n", desc);

  unsigned int bytes = n * sizeof(float);

  // peform kernel operation on h_a to make sure it's on the GPU
  do_stuff<<<(n + 255)/256 /*blocks*/, 256/*threads*/>>>(d, n);

  // events for timing
  hipEvent_t startEvent, stopEvent; 

  checkCuda( hipEventCreate(&startEvent) );
  checkCuda( hipEventCreate(&stopEvent) );

  checkCuda( hipEventRecord(startEvent, 0) );
  checkCuda( hipMemcpy(d, h_a, bytes, hipMemcpyHostToDevice) );
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );

  float time;
  checkCuda( hipEventElapsedTime(&time, startEvent, stopEvent) );
  printf("  Host to Device bandwidth (GB/s): %f\n", bytes * 1e-6 / time);

  checkCuda( hipEventRecord(startEvent, 0) );
  checkCuda( hipMemcpy(h_b, d, bytes, hipMemcpyDeviceToHost) );
  checkCuda( hipEventRecord(stopEvent, 0) );
  checkCuda( hipEventSynchronize(stopEvent) );

  checkCuda( hipEventElapsedTime(&time, startEvent, stopEvent) );
  printf("  Device to Host bandwidth (GB/s): %f\n", bytes * 1e-6 / time);

  for (unsigned i = 0; i < n; ++i) {
    if (h_a[i] != h_b[i]) {
      printf("*** %s transfers failed ***\n", desc);
      break;
    }
  }

  // clean up events
  checkCuda( hipEventDestroy(startEvent) );
  checkCuda( hipEventDestroy(stopEvent) );
}

int main()
{
  unsigned int nElements = 4*1024*1024;
  const unsigned int bytes = nElements * sizeof(float);

  // host arrays
  float *h_aPageable, *h_bPageable;   
  float *h_aPinned, *h_bPinned;
  float *h_aManaged, *h_bManaged;

  // device array
  float *d_a;

  // allocate and initialize
  h_aPageable = (float*)malloc(bytes);                    // host pageable
  h_bPageable = (float*)malloc(bytes);                    // host pageable
  checkCuda( hipHostMalloc((void**)&h_aPinned, bytes, hipHostMallocDefault) ); // host pinned
  checkCuda( hipHostMalloc((void**)&h_bPinned, bytes, hipHostMallocDefault) ); // host pinned
  checkCuda( hipMallocManaged((void **) &h_aManaged, bytes) );
  checkCuda( hipMallocManaged((void **) &h_bManaged, bytes) );
  checkCuda( hipMalloc((void**)&d_a, bytes) );           // device

  for (unsigned i = 0; i < nElements; ++i) h_aPageable[i] = i;      
  memcpy(h_aPinned, h_aPageable, bytes);
  memcpy(h_aPinned, h_aManaged, bytes);
  memset(h_bPageable, 0, bytes);
  memset(h_bPinned, 0, bytes);
  memset(h_bManaged, 0, bytes);

  // output device info and transfer size
  hipDeviceProp_t prop;
  checkCuda( hipGetDeviceProperties(&prop, 0) );

  printf("\nDevice: %s\n", prop.name);
  printf("Transfer size (MB): %d\n", bytes / (1024 * 1024));

  // perform copies and report bandwidth
  profileCopies(h_aPageable, h_bPageable, d_a, nElements, "Pageable");
  profileCopies(h_aPinned, h_bPinned, d_a, nElements, "Pinned");
  profileCopies(h_aManaged, h_bManaged, d_a, nElements, "Managed");

  printf("n");

  // cleanup
  hipFree(d_a);
  hipHostFree(h_aPinned);
  hipHostFree(h_bPinned);
  free(h_aPageable);
  free(h_bPageable);

  return 0;
}
