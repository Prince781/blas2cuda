#include "blas2cuda.h"
#include <stdio.h>
#include <stdlib.h>

hipblasHandle_t b2c_handle;

struct options b2c_options = { false };

static void set_options(void) {
    /* TODO: use secure_getenv() ? */
    char *options = getenv("BLAS2CUDA_OPTIONS");
    char *saveptr = NULL;
    char *option = NULL;

    option = strtok_r(options, ",", &saveptr);
    while (option != NULL) {
        if (strcmp(option, "debug") == 0)
            b2c_options.debug = true;
        option = strtok_r(NULL, ",", &saveptr);
    }
}

void *b2c_copy_to_gpu(const void *devbuf, size_t size)
{
    void *gpubuf = NULL;

    hipMalloc(&gpubuf, size);

    if (gpubuf == NULL)
        return gpubuf;

    hipMemcpy(gpubuf, devbuf, size, hipMemcpyHostToDevice);

    return gpubuf;
}

void *b2c_copy_to_cpu(const void *gpubuf, size_t size)
{
    void *devbuf = NULL;

    devbuf = malloc(size);

    if (devbuf == NULL)
        return devbuf;

    hipMemcpy(devbuf, gpubuf, size, hipMemcpyDeviceToHost);

    return devbuf;
}

__attribute__((constructor))
void blas2cuda_init(void)
{
    switch (hipblasCreate(&b2c_handle)) {
        case HIPBLAS_STATUS_SUCCESS:
            /* do nothing */
            break;
        case HIPBLAS_STATUS_ALLOC_FAILED:
            fprintf(stderr, "cuBLAS: failed to allocate resources\n");
        case HIPBLAS_STATUS_NOT_INITIALIZED:
        default:
            fprintf(stderr, "cuBLAS: failed to initialize cuBLAS\n");
            exit(EXIT_FAILURE);
            break;
    }

    set_options();
}

__attribute__((destructor))
void blas2cuda_fini(void)
{
    if (hipblasDestroy(b2c_handle) == HIPBLAS_STATUS_NOT_INITIALIZED)
        fprintf(stderr, "cuBLAS: failed to destroy. Not initialized\n");
}
