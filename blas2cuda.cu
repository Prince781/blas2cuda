#include "blas2cuda.h"
#include <stdio.h>

hipblasHandle_t b2c_handle;

__attribute__((constructor))
void blas2cuda_init(void)
{
    switch (hipblasCreate(&b2c_handle)) {
        case HIPBLAS_STATUS_SUCCESS:
            /* do nothing */
            break;
        case HIPBLAS_STATUS_ALLOC_FAILED:
            fprintf(stderr, "cuBLAS: failed to allocate resources\n");
        case HIPBLAS_STATUS_NOT_INITIALIZED:
        default:
            fprintf(stderr, "cuBLAS: failed to initialize cuBLAS\n");
            exit(EXIT_FAILURE);
            break;
    }
}

__attribute__((destructor))
void blas2cuda_fini(void)
{
    if (hipblasDestroy(b2c_handle) == HIPBLAS_STATUS_NOT_INITIALIZED)
        fprintf(stderr, "cuBLAS: failed to destroy. Not initialized\n");
}
