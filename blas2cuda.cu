#include "blas2cuda.h"
#include "lib/obj_tracker.h"
#include <stdio.h>
#include <stdlib.h>

static bool init = false;

hipblasHandle_t b2c_handle;

struct options b2c_options = { false, false, false };

static void set_options(void) {
    /* TODO: use secure_getenv() ? */
    char *options = getenv("BLAS2CUDA_OPTIONS");
    char *saveptr = NULL;
    char *option = NULL;
    bool help = false;

    if (!options)
        return;

    option = strtok_r(options, ";", &saveptr);
    while (option != NULL) {
        if (strcmp(option, "help") == 0) {
            if (!help) {
                fprintf(stderr, 
                        "blas2cuda options:\n"
                        "You can chain these options with a semicolon (;)\n"
                        "help           -- print help\n"
                        "debug_execfail -- debug kernel failures\n"
                        "debug_exec     -- debug kernel invocations\n"
                        "trace_copy     -- trace copies between CPU and GPU\n"
                       );
                help = true;
            }
        } 
        else if (strcmp(option, "debug_execfail") == 0)
            b2c_options.debug_execfail = true;
        else if (strcmp(option, "debug_exec") == 0)
            b2c_options.debug_exec = true;
        else if (strcmp(option, "trace_copy") == 0)
            b2c_options.trace_copy = true;
        else {
            fprintf(stderr, "blas2cuda: unknown option '%s'. Set BLAS2CUDA_OPTIONS=help.\n", option);
        }
        option = strtok_r(NULL, ",", &saveptr);
    }
}

void init_cublas(void) {
    if (!init) {
        switch (hipblasCreate(&b2c_handle)) {
            case HIPBLAS_STATUS_SUCCESS:
                /* do nothing */
                break;
            case HIPBLAS_STATUS_ALLOC_FAILED:
                fprintf(stderr, "blas2cuda: failed to allocate resources\n");
            case HIPBLAS_STATUS_NOT_INITIALIZED:
            default:
                fprintf(stderr, "blas2cuda: failed to initialize cuBLAS\n");
                exit(EXIT_FAILURE);
                break;
        }
        obj_tracker_init();
        init = true;
    }
}

void *b2c_copy_to_gpu(const void *devbuf, size_t size)
{
    void *gpubuf = NULL;

    init_cublas();

    hipMalloc(&gpubuf, size);

    if (!gpubuf)
        return NULL;

    hipMemcpy(gpubuf, devbuf, size, hipMemcpyHostToDevice);

    if (b2c_options.trace_copy)
        printf("%s: %zu B : CPU ---> GPU\n", __func__, size);

    return gpubuf;
}

void *b2c_copy_to_cpu(const void *gpubuf, size_t size)
{
    void *devbuf = NULL;

    init_cublas();

    devbuf = malloc(size);

    if (devbuf == NULL)
        return devbuf;

    hipMemcpy(devbuf, gpubuf, size, hipMemcpyDeviceToHost);

    if (b2c_options.trace_copy)
        printf("%s: %zu B : GPU ---> CPU\n", __func__, size);

    return devbuf;
}

void b2c_copy_from_gpu(void *cpubuf, const void *gpubuf, size_t size)
{
    hipMemcpy(cpubuf, gpubuf, size, hipMemcpyDeviceToHost);

    if (b2c_options.trace_copy)
        printf("%s: %zu B : GPU ---> CPU\n", __func__, size);
}

__attribute__((constructor))
void blas2cuda_init(void)
{
    set_options();
    printf("initialized blas2cuda\n");
}

__attribute__((destructor))
void blas2cuda_fini(void)
{
    if (init && hipblasDestroy(b2c_handle) == HIPBLAS_STATUS_NOT_INITIALIZED)
        fprintf(stderr, "blas2cuda: failed to destroy. Not initialized\n");
    printf("decommissioned blas2cuda\n");
    obj_tracker_fini();
}
